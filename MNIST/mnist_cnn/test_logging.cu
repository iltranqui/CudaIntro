
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <iomanip>

int main() {
    std::cout << "MNIST CNN Logging Test" << std::endl;
    
    // Create a log file
    std::ofstream log_file("mnist_cnn_test.log");
    if (log_file.is_open()) {
        log_file << "epoch,train_loss,train_accuracy,val_loss,val_accuracy,time_seconds" << std::endl;
        
        // Simulate training data
        for (int epoch = 1; epoch <= 10; epoch++) {
            float train_loss = 2.5f / epoch;
            float train_accuracy = 0.5f + 0.05f * epoch;
            float val_loss = 2.0f / epoch;
            float val_accuracy = 0.55f + 0.04f * epoch;
            float time_seconds = 10.0f + epoch;
            
            log_file << epoch << ","
                    << std::fixed << std::setprecision(4) << train_loss << ","
                    << std::fixed << std::setprecision(4) << train_accuracy << ","
                    << std::fixed << std::setprecision(4) << val_loss << ","
                    << std::fixed << std::setprecision(4) << val_accuracy << ","
                    << std::fixed << std::setprecision(2) << time_seconds << std::endl;
            
            std::cout << "Epoch " << epoch << "/10"
                      << ", Train Loss: " << std::fixed << std::setprecision(4) << train_loss
                      << ", Train Acc: " << std::fixed << std::setprecision(4) << train_accuracy
                      << ", Val Loss: " << std::fixed << std::setprecision(4) << val_loss
                      << ", Val Acc: " << std::fixed << std::setprecision(4) << val_accuracy
                      << ", Time: " << std::fixed << std::setprecision(2) << time_seconds << "s"
                      << std::endl;
        }
        
        log_file.close();
        std::cout << "Training data logged to mnist_cnn_test.log" << std::endl;
    } else {
        std::cerr << "Could not open log file for writing." << std::endl;
    }
    
    return 0;
}
