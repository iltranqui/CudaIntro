#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <stdio.h>
#include "main_header.cuh"

#define CONV1D

int main() {

	// AI conv deform - Deformable Convolution with Bilinear Interpolation
	conv2d_deform_infer();
	conv2d_deform_training_loop();

	// Other convolution implementations
	// conv 2d pmpp
	//launchConvolution2DBasicKernel();
	// conv 2d backpass
	//conv2d_backpass();
	// conv_1d_host: CPU vs GPU
	//conv_1d_host();
	// conv_1d_padded: CPU vs GPU
	//conv_1d_padded();
	//conv_1d_padded_shared();
	//conv_1d_cudnn();
	// benchmarkconv 2d
	//benchmark_convolution2d();

	// cudnn loop
	//cudnn_loop();

	// max pooling 1d
	//max_pooling_1d();
	//average_pooling_1d();

	return 0;
}
