#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <random>

// https://tensara.org/problems/max-pool-1d/

// CUDA error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)



// CUDA kernel performing 1D max pooling
__global__ void maxpool1d_kernel(const float* input, int kernel_size, int stride, int padding, int dilation, float* output, size_t H, int H_out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < H_out) {
        // Start with negative infinity for the max value
        float max_val = -INFINITY;
        for (int m = 0; m < kernel_size; m++) {
            // Calculate the input index for this kernel element
            int index = stride * i + dilation * m - padding;
            // If the index is out-of-bound, treat the value as -infinity
            float value = (index < 0 || index >= H) ? -INFINITY : input[index];
            max_val = fmaxf(max_val, value);
        }
        output[i] = max_val;
    }
}

// Note: input and output are device pointers to float arrays
extern "C" void solution_max_pooling_1d(const float* input, int kernel_size, int stride, int padding, int dilation, float* output, size_t H) {
    // Calculate output size using:
    // H_out = floor((H + 2*padding - dilation*(kernel_size-1) - 1) / stride) + 1
    int H_out = ((H + 2 * padding - dilation * (kernel_size - 1) - 1) / stride) + 1;

    // Set up grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (H_out + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the CUDA kernel
    maxpool1d_kernel << <blocksPerGrid, threadsPerBlock >> > (input, kernel_size, stride, padding, dilation, output, H, H_out);

    // Wait for the kernel to finish before returning
    hipDeviceSynchronize();
}

void generate_random_data(std::vector<float>& data) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);
    for (auto& val : data) {
        val = dist(gen);
    }
}

int max_pooling_1d() {
    size_t H = 16;  // Length of the 1D input tensor
    int kernel_size = 3;
    int stride = 1;
    int padding = 1;
    int dilation = 1;

    std::vector<float> h_input(H);
    generate_random_data(h_input);

    float* d_input, * d_output;
    CUDA_CHECK(hipMalloc(&d_input, H * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, H * sizeof(float))); // Assuming output is same size
    CUDA_CHECK(hipMemcpy(d_input, h_input.data(), H * sizeof(float), hipMemcpyHostToDevice));

    // Call the CUDA function
    solution_max_pooling_1d(d_input, kernel_size, stride, padding, dilation, d_output, H);

    // Copy result back to host
    std::vector<float> h_output(H);
    CUDA_CHECK(hipMemcpy(h_output.data(), d_output, H * sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "Input: ";
    for (float val : h_input) std::cout << val << " ";
    std::cout << "\nOutput: ";
    for (float val : h_output) std::cout << val << " ";
    std::cout << std::endl;

    // Free device memory
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));

    return 0;
}