#include "hip/hip_runtime.h"
﻿#define EXECUTION_PARAMETERS  // define this macro to include the execution parameters in the output


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>
#include <random>
#include <stdexcept>
#include <typeinfo>

// Template function to create and populate a 2D matrix with random values
/*
 * @param width: width of the matrix
 * @param height: height of the matrix
 * @param min_value: minimum value for the random number generator
 * @param max_value: maximum value for the random number generator
 * @return: pointer to the 2D matrix
 */

template <typename T>
T** create2DMatrix(size_t width, size_t height, T min_value, T max_value) {
    // Random number generator
	std::random_device rd;     // Obtain a random number from hardware
	std::mt19937 gen(rd());    // Seed the generator
	std::uniform_real_distribution<> dis(min_value, max_value);   // Define the range of the random number generator

    /*  // need to figure out what this does 
    if constexpr (std::is_integral<T>::value) {
        std::uniform_int_distribution<> int_dis(min_value, max_value);
        dis = int_dis; // Switch to integer distribution for integral types
    }
	*/

    // Allocate memory for the matrix
    T** matrix = new T * [height];
    for (size_t i = 0; i < height; ++i) {
        matrix[i] = new T[width];
        for (size_t j = 0; j < width; ++j) {
            matrix[i][j] = dis(gen); // Assign random values
        }
    }
    return matrix;
}

// Function to create a 2D matrix based on type_def
/*
 * @param width: width of the matrix
 * @param height: height of the matrix
 * @param type_def: type of the matrix (int, float, double)
 * @param min_value: minimum value for the random number generator
 * @param max_value: maximum value for the random number generator
 * @return: void pointer to the 2D matrix
 */

void* createMatrix(size_t width, size_t height, const std::string& type_def, double min_value, double max_value) {
    if (type_def == "int") {
        return static_cast<void*>(create2DMatrix<int>(width, height, static_cast<int>(min_value), static_cast<int>(max_value)));
        // The static_cast performs the conversion at compile time
    }
    else if (type_def == "float") {
        return static_cast<void*>(create2DMatrix<float>(width, height, static_cast<float>(min_value), static_cast<float>(max_value)));
    }
    else if (type_def == "double") {
        return static_cast<void*>(create2DMatrix<double>(width, height, min_value, max_value));
    }
    else {
        throw std::invalid_argument("Unsupported type: " + type_def);
    }
}

// Helper function to print an int matrix (for demonstration purposes)ù
/*
* @param matrix: pointer to the 2D matrix
* @param width: width of the matrix
* @param height: height of the matrix
*/
template <typename T>
void printMatrix(T** matrix, size_t width, size_t height) {
    for (size_t i = 0; i < height; ++i) {
        for (size_t j = 0; j < width; ++j) {
            std::cout << matrix[i][j] << " ";
        }
        std::cout << std::endl;
    }
}

// Kernel function to add two matrices
__global__ void addMatrices(const int* matrix_1, const int* matrix_2, int* matrix_result, size_t width, size_t height) {
    // Calculate the global thread ID for the 2D grid
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform addition only for valid indices
    if (row < height && col < width) {
        size_t idx = row * width + col; // Flattened index
        matrix_result[idx] = matrix_1[idx] + matrix_2[idx];
    }
}

// How to use the createMatrix function
// Create a 2D matrix with random values
//int** matrix = static_cast<int**>(createMatrix(width, height, type_def, min_value, max_value));

int main() {
    // Matrix dimensions
    size_t width = 32;
    size_t height = 32;
    // Type of the matrix
    std::string type_def = "int";
    // Random number generator limits
    double min_value = 0;
    double max_value = 10;
    // Create a 2D matrix with random values
    int** matrix_1 = static_cast<int**>(createMatrix(width, height, type_def, min_value, max_value));
    int** matrix_2 = static_cast<int**>(createMatrix(width, height, type_def, min_value, max_value));

    int** matrix_result = new int* [height];
    // Print the matrix
    //printMatrix(matrix, width, height);

    // allocate memory in GPU
    int* d_matrix_1;
    int* d_matrix_2;
    int* d_matrix_result;
    hipMalloc(&d_matrix_1, width * height * sizeof(int));
    hipMalloc(&d_matrix_2, width * height * sizeof(int));
    hipMalloc(&d_matrix_result, width * height * sizeof(int));

    // copy data to GPU
    hipMemcpy(d_matrix_1, matrix_1[0], width * height * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix_2, matrix_2[0], width * height * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threadsPerBlock(16, 16); // 16x16 threads per block
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the kernel
    addMatrices << <blocksPerGrid, threadsPerBlock >> > (d_matrix_1, d_matrix_2, d_matrix_result, width, height);

    // Copy the result back to the host
    hipMemcpy(matrix_result[0], d_matrix_result, width * height * sizeof(int), hipMemcpyDeviceToHost);

    // Print the resulting matrix
    std::cout << "Resultant Matrix: " << std::endl;
    for (size_t i = 0; i < height; i++) {
        for (size_t j = 0; j < width; j++) {
            std::cout << matrix_result[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Free GPU memory
    hipFree(d_matrix_1);
    hipFree(d_matrix_2);
    hipFree(d_matrix_result);

    // Free host memory
    delete[] matrix_1[0];
    delete[] matrix_1;
    delete[] matrix_2[0];
    delete[] matrix_2;
    delete[] matrix_result[0];
    delete[] matrix_result;

    return 1;


}